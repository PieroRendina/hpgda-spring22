#include "hip/hip_runtime.h"
// Copyright (c) 2020, 2021, NECSTLab, Politecnico di Milano. All rights reserved.

// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NECSTLab nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//  * Neither the name of Politecnico di Milano nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.

// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include <sstream>
#include "personalized_pagerank.cuh"
#include <list>
#include <vector>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
namespace chrono = std::chrono;
using clock_type = chrono::high_resolution_clock;

//////////////////////////////
//////////////////////////////
#define CHECK_KERNELCALL()                                                                \
    {                                                                                     \
        const hipError_t err = hipGetLastError();                                       \
        if (err != hipSuccess)                                                           \
        {                                                                                 \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }

#define CHECK(call)                                                                       \
    {                                                                                     \
        const hipError_t err = call;                                                     \
        if (err != hipSuccess)                                                           \
        {                                                                                 \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }

// Write GPU kernel here!

// Change a value of an array
__global__ void modify_device_array_value(double *device_array, int index, double value)
{
    // residues[personalization_vertex] = 1.0;
    device_array[index] = value;
}

// Update reserve and residue
__global__ void update_pi0_and_r(int *frontier_d, double alpha, double *pi0_d, double *r_d, int dim_frontier)
{
    // compute the index of the vertex in the frontier
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < dim_frontier; i += blockDim.x * gridDim.x)
    {
        pi0_d[frontier_d[i]] += alpha * r_d[frontier_d[i]];
        r_d[frontier_d[i]] = 0.0;
    }
}

// Update the residues and the frontier
__global__ void compute_new_frontier(double *r_d, double rmax, bool *flags_d, int *outdegrees, double alpha, int *out_neighbors, int tot_neighbors, int *frontier, int dim_frontier)
{
    /*for(int j = blockIdx.x * blockDim.x + threadIdx.x; j < tot_neighbors ; j += blockDim.x * gridDim.x) {
        if(outdegrees[j] > 0){
            r_d[out_neighbors[j]] += (1 - alpha)*r_d[out_neighbors[j]]/outdegrees[j];
            if(r_d[out_neighbors[j]]/outdegrees[j] > rmax && flags_d[out_neighbors[j]] != true) {
                //thread syncing before updating the flags
                __syncthreads();
                flags_d[out_neighbors[j]] = true;
          }
        }else{
            r_d[out_neighbors[j]] = 0;
        }
    }*/
    for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < dim_frontier; j += blockDim.x)
    {
        int neighbor_idx = 0;
        if (j - 1 >= 0)
        {
            neighbor_idx += outdegrees[frontier[j - 1]];
        }
        int idx = neighbor_idx;
        while (idx < neighbor_idx + outdegrees[frontier[j]])
        {
            // changed out_neighbors[j] in frontier[j]
            r_d[out_neighbors[idx]] += (1 - alpha) * r_d[frontier[j]] / outdegrees[frontier[j]];
            // printf("residue considered = %lf\n", r_d[out_neighbors[idx]]);
            if (outdegrees[out_neighbors[idx]] > 0)
            {
                if (r_d[out_neighbors[idx]] / outdegrees[out_neighbors[idx]] > rmax && flags_d[out_neighbors[idx]] != true)
                {
                    __syncthreads();
                    flags_d[out_neighbors[idx]] = true;
                }
            }
            idx++;
        }
    }
}

// Monte Carlo sampling
__global__ void random_walks(double rsum, double w, double *pi0_d, int tot_nodes)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    bool stop = false;
    hiprandState state;
    hiprand_init(0, i, 0, &state);
    float result = hiprand_uniform(&state);
    if (result > 0.15)
    {
        stop = true;
    }
    printf("tid = %d, result = %f\n", i, result);
    for (; i < tot_nodes; i += blockDim.x * gridDim.x)
    {

        /*
        double wi = ceil(residues_d[i] * w / rsum);
        printf("wi = %lf at iteration %d", wi, i);
        double ai = (residues_d[i] * w) / (rsum * wi);

        for (int j = 0; j < wi; j++)
        {
            // compute monte carlo
            // pi0_d[i] += ai*rsum/w
        }
        */
    }
}

//////////////////////////////
/* CPU function */
//////////////////////////////

void PersonalizedPageRank::initialize_outdegrees()
{
    for (int i = 0; i < V - 1; i++)
    {
        // printf("node in the frontier = %d\n", frontier[i]);
        int start_idx = neighbor_start_idx[i];
        int end_idx = neighbor_start_idx[i + 1];
        // outdegree computation
        outdegrees[i] = end_idx - start_idx;
        // the node is dropped from the frontier
        flags[frontier[i]] = false;
    }
    flags[frontier[V]] = false;
}

void PersonalizedPageRank::update_frontiers()
{

    /* --- Compute the number of neighbors and drop the nodes from the frontier --- */
    // allocate the vector to store the degree of each node in the frontier
    int tot_neighbors = 0;
    // int *outdegrees;
    /*
    one outdegree for each member of the frontier
    err = hipMallocManaged(&outdegrees, sizeof(int) * dim_frontier);
    printf("\nDim frontier = %d\n", dim_frontier);
    */
    for (int i = 0; i < dim_frontier; i++)
    {

        tot_neighbors += outdegrees[frontier[i]];
        // the node is dropped from the frontier
        flags[frontier[i]] = false;
    }
    // printf("Num outneighbors = %d\n", tot_neighbors);
    /* --- Add the neighbours to be considered in the vector out_neighbors --- */
    int *out_neighbors;
    /* all the neighbors to be considered */
    err = hipMallocManaged(&out_neighbors, sizeof(int) * tot_neighbors);
    int counter = 0;
    for (int i = 0; i < dim_frontier; i++)
    {
        int start_idx = neighbor_start_idx[frontier[i]];
        int end_idx = neighbor_start_idx[frontier[i] + 1];
        for (int j = start_idx; j < end_idx; j++)
        {
            out_neighbors[counter] = neighbors[j];
            counter += 1;
        }
    }
    /* --- Update of the frontier --- */
    // int new_frontier_dim = 3*dim_frontier;
    // int new_frontier_dim = 30 * dim_frontier;
    // int *new_frontier;
    /* --- For simplicity I allocate a frontier that is long the number of vertex --- */
    err = hipMallocManaged(&new_frontier, sizeof(int) * V);
    // int *new_frontier = (int *)malloc(new_frontier_dim * sizeof(int));

    /* For each out_neighbor updates the residue and checks if it has to be added to the frontier */
    // compute_new_frontier<<<ceil(tot_neighbors/1024)+1, ceil(tot_neighbors/ceil(tot_neighbors/1024))+1>>>(residues_d, rmax, flags_d, outdegrees, alpha, out_neighbors, tot_neighbors);

    int n_blocks = ceil(tot_neighbors / 1024) + 1;
    int n_threads = ceil(tot_neighbors / n_blocks) + 1;
    hipMemcpy(flags_d, flags, sizeof(bool) * V, hipMemcpyHostToDevice);
    compute_new_frontier<<<1, 1>>>(residues_d, rmax, flags_d, outdegrees, alpha, out_neighbors, tot_neighbors, frontier, dim_frontier);
    CHECK(hipDeviceSynchronize());

    hipMemcpy(flags, flags_d, sizeof(bool) * V, hipMemcpyDeviceToHost);

    // Compute new frontier
    int idx_frontier = 0;
    for (int i = 0; i < V; i++)
    {
        if (flags[i] == true)
        {
            /*
            if(idx_frontier >= new_frontier_dim){
              new_frontier_dim = 3 * new_frontier_dim;
              new_frontier = (int *)realloc(new_frontier, new_frontier_dim * sizeof(int));
              printf("--- realloc done --- \n");
            }*/
            new_frontier[idx_frontier] = i;
            idx_frontier++;
        }
    }

    /* -- Check the flag -- */

    // hipFree(frontier);
    // frontier = new_frontier;
    new_dim_frontier = idx_frontier;
    /*std::cout << "----- Updated frontier -----\n";
    for (int i = 0; i < new_dim_frontier; i++)
    {
        std::cout << new_frontier[i] << " ";
    }*/
}

__global__ void initialize_csr_parallel(int *x, int *y, int V, int E, int **out_neighbors, int *outdegrees)
{
    for (int index = threadIdx.x + blockDim.x * blockIdx.x; index < V; index += blockDim.x)
    {
        int n_neighbors = 0;
        for (int j = 0; j < E; j++)
        {
            if (y[j] == index)
            {
                n_neighbors++;
            }
        }
        out_neighbors[index] = (int *)malloc(sizeof(int) * n_neighbors);
        outdegrees[index] = n_neighbors;
        int curr_idx = 0;
        for (int j = 0; j < E; j++)
        {
            if (y[j] == index)
            {
                out_neighbors[index][curr_idx] = x[j];
                curr_idx++;
            }
        }
    }
}

__global__ void print_neighbors(int **out_neighbors, int *outdegrees, int V)
{
    for (int i = 0; i < V; i++)
    {
        for (int j = 0; j < outdegrees[i]; j++)
        {
            printf("%d ", out_neighbors[i][j]);
        }
    }
}

// Construct CSR representation of the graph
void PersonalizedPageRank::initialize_csr()
{
    // allocate a vector containing the index of the starting neighbor
    neighbor_start_idx = (int *)malloc((V + 1) * sizeof(int));
    neighbors = (int *)malloc(E * sizeof(int));

    int curr_neighbor = 0;
    int curr_neighbor_start_idx = 1;
    neighbor_start_idx[0] = 0;

    for (int i = 0; i < V; i++)
    {
        // auto start = std::chrono::system_clock::now();
        for (int j = 0; j < E; j++)
        {
            if (y[j] == i)
            {
                neighbors[curr_neighbor] = x[j];
                curr_neighbor++;
            }
        }
        neighbor_start_idx[curr_neighbor_start_idx] = curr_neighbor;
        // outdegrees[i] = neighbor_start_idx[curr_neighbor_start_idx] - neighbor_start_idx[curr_neighbor_start_idx - 1];
        curr_neighbor_start_idx++;
        /*auto end = std::chrono::system_clock::now();
        std::chrono::duration<double> elapsed_seconds = end-start;
        printf("Iteration done in %f\n", elapsed_seconds.count());*/
        
    }
    initialize_outdegrees();
/*
    printf("\n --- Finished CSR initialization --- \n");

    std::cout << "----- Outdegrees -----\n";
        for (int i = 0; i < V ; i++)
        {
            std::cout << outdegrees[i] << " ";
        }
        std::cout << "\n----- Neighbours -----\n";
        for (int i = 0; i < V + 1; i++)
        {
            std::cout << neighbor_start_idx[i] << " ";
        }
    
    std::cout << "\n----- Neighbours -----\n";
    for (int i = 0; i < E; i++)
    {
        std::cout << neighbors[i] << " ";
    }*/
}

// CPU Utility functions;

// Read the input graph and initialize it;
void PersonalizedPageRank::initialize_graph() {
    // Read the graph from an MTX file;
    int num_rows = 0;
    int num_columns = 0;
    read_mtx(graph_file_path.c_str(), &x, &y, &val,
        &num_rows, &num_columns, &E, // Store the number of vertices (row and columns must be the same value), and edges;
        true,                        // If true, read edges TRANSPOSED, i.e. edge (2, 3) is loaded as (3, 2). We set this true as it simplifies the PPR computation;
        false,                       // If true, read the third column of the matrix file. If false, set all values to 1 (this is what you want when reading a graph topology);
        debug,                 
        false,                       // MTX files use indices starting from 1. If for whatever reason your MTX files uses indices that start from 0, set zero_indexed_file=true;
        true                         // If true, sort the edges in (x, y) order. If you have a sorted MTX file, turn this to false to make loading faster;
    );
    if (num_rows != num_columns) {
        if (debug) std::cout << "error, the matrix is not squared, rows=" << num_rows << ", columns=" << num_columns << std::endl;
        exit(-1);
    } else {
        V = num_rows;
    }
    if (debug) std::cout << "loaded graph, |V|=" << V << ", |E|=" << E << std::endl;

    // Compute the dangling vector. A vertex is not dangling if it has at least 1 outgoing edge;
    dangling.resize(V);
    std::fill(dangling.begin(), dangling.end(), 1);  // Initially assume all vertices to be dangling;
    for (int i = 0; i < E; i++) {
        // Ignore self-loops, a vertex is still dangling if it has only self-loops;
        if (x[i] != y[i]) dangling[y[i]] = 0;
    }
    // Initialize the CPU PageRank vector;
    pr.resize(V);
    pr_golden.resize(V);
    // Initialize the value vector of the graph (1 / outdegree of each vertex).
    // Count how many edges start in each vertex (here, the source vertex is y as the matrix is transposed);
    int *outdegree = (int *) calloc(V, sizeof(int));
    for (int i = 0; i < E; i++) {
        outdegree[y[i]]++;
    }
    // Divide each edge value by the outdegree of the source vertex;
    for (int i = 0; i < E; i++) {
        val[i] = 1.0 / outdegree[y[i]];  
    }
    free(outdegree);
}

//////////////////////////////
//////////////////////////////

// Allocate data on the CPU and GPU;
void PersonalizedPageRank::alloc()
{

    // Load the input graph and preprocess it;
    initialize_graph();
    
    // allocate the mask to store the status of the nodes in the frontier (all false by default)
    flags = (bool *)calloc(V, sizeof(bool));
    // at the beginning the frontier contains just the personalization vertex
    // frontier = (int *)malloc(sizeof(int));
    err = hipMallocManaged(&frontier, sizeof(int));
    err = hipMallocManaged(&outdegrees, sizeof(int) * V);

    /*
    initialize_csr_parallel<<<1,16>>>(x_d, y_d, V, E, out_neighbors, outdegrees);
    CHECK(hipDeviceSynchronize());
    print_neighbors<<<1,1>>>(out_neighbors, outdegrees, V);
    CHECK(hipDeviceSynchronize());
    printf("\n --- Finished parallel graph initialization --- \n");
    */
    // finish attempt
    

    initialize_csr();
    // Allocate any GPU data here;
    // TODO!

    // CSR variables
    err = hipMalloc(&neighbor_start_idx_d, sizeof(int) * (V + 1));
    err = hipMalloc(&neighbors_d, sizeof(int) * E);

    // Forward push variables
    err = hipMalloc(&pi0_d, sizeof(double) * V);
    err = hipMalloc(&residues_d, sizeof(double) * V);
    err = hipMalloc(&flags_d, sizeof(bool) * V);
    // err = hipMalloc(&frontier_d, sizeof(int));
    //  some variables may be missing

    /*double * personal_x = (double*)malloc(sizeof(double)*V);
    hipMemcpy(personal_x, residues_d, sizeof(double)*V, hipMemcpyDeviceToHost);
    std::cout << "\nValue of x = " << personal_x[0];*/

    
}

// Initialize data;
void PersonalizedPageRank::init()
{
    // Do any additional CPU or GPU setup here;
    // TODO!

    // Compute Rmax
    threshold = 1.0 / V; // should be O(1/n) but i don't know yet which is the best value
    rmax = (convergence_threshold / sqrt(E)) * sqrt(threshold / (((2.0 * convergence_threshold / 3.0) + 2.0) * (log(2.0 / failure_probability))));
    // std::cout << "rmax = " << rmax << '\n'; // It seems really small

    hipMemcpy(neighbor_start_idx_d, neighbor_start_idx, sizeof(int) * (V + 1), hipMemcpyHostToDevice);
    hipMemcpy(neighbors_d, neighbors, sizeof(int) * E, hipMemcpyHostToDevice);
}

// Reset the state of the computation after every iteration.
// Reset the result, and transfer data to the GPU if necessary;
void PersonalizedPageRank::reset()
{
    // Reset the PageRank vector (uniform initialization, 1 / V for each vertex);
    std::fill(pr.begin(), pr.end(), 1.0 / V);
    // Generate a new personalization vertex for this iteration;
    personalization_vertex = rand() % V;
    if (debug)
        std::cout << "personalization vertex=" << personalization_vertex << std::endl;

    // Do any GPU reset here, and also transfer data to the GPU;
    // TODO!

    hipMemset(residues_d, 0.0, sizeof(double) * V);
    hipMemset(pi0_d, 0.0, sizeof(double) * V);
    hipMemcpy(flags_d, flags, sizeof(bool) * V, hipMemcpyHostToDevice);

    /*
    // Free old variables
    free(positive_residues);
    free(new_frontier);
    free(frontier);
    free(residues);
    */
}

void PersonalizedPageRank::execute(int iter)
{
    // Do the GPU computation here, and also transfer results to the CPU;

    // ---> START FORWARD PUSH

    // Initialize the frontier
    dim_frontier = 1;
    frontier[0] = personalization_vertex;
    flags[personalization_vertex] = true;

    // Set the residue of the source of the PPR algorithm to 1
    modify_device_array_value<<<1, 1>>>(residues_d, personalization_vertex, 1.0);
    CHECK(hipDeviceSynchronize());

    while (dim_frontier > 0)
    {
        // Update the residues and decide whether the visited node has to be added to the frontier
        update_frontiers();
        CHECK(hipDeviceSynchronize());

        // Update pi0 and the residues of the node in the frontier yet
        int n_blocks = ceil(dim_frontier / 1024) + 1;
        int n_threads = ceil(dim_frontier / n_blocks) + 1;
        update_pi0_and_r<<<n_blocks, n_threads>>>(frontier, alpha, pi0_d, residues_d, dim_frontier);
        CHECK(hipDeviceSynchronize());

        frontier = new_frontier;
        dim_frontier = new_dim_frontier;
    }
    // END FORWARD PUSH <---

    // ---> START RANDOM SAMPLING (MONTECARLO)

    residues = (double *)malloc(sizeof(double) * V);
    hipMemcpy(residues, residues_d, sizeof(double) * V, hipMemcpyDeviceToHost);

    // Counter for storing the number of nodes with positive residue
    int count_positive_residues = 0;

    // Structure for storing the nodes with positive residue
    positive_residues = (double *)malloc(sizeof(int) * V);
    rsum = 0;
    // Compute rsum and save in an array all the nodes with positive residues
    for (int i = 0; i < V; i++)
    {
        if (residues[i] > 0)
        {
            rsum += residues[i];
            positive_residues[count_positive_residues] = i;
            count_positive_residues++;
        }
    }

    // Adjust the size
    positive_residues = (double *)realloc(positive_residues, count_positive_residues);

    // Compute w
    w = rsum * ((2 * convergence_threshold / 3 + 2) * log(2.0 / failure_probability) / (convergence_threshold * convergence_threshold * threshold));

    // For every node in positive_residue instantiate a thread on the GPU and compute the random walks
    int n_blocks = ceil(count_positive_residues / 1024) + 1;
    int n_threads = ceil(count_positive_residues / n_blocks) + 1;
    random_walks<<<n_blocks, n_threads>>>(rsum, w, pi0_d, count_positive_residues);

    // END RANDOM SAMPLING (MONTECARLO) <---

    // Save the GPU PPR values into the "pr" array
    pi0 = (double *)malloc(sizeof(double) * V);
    hipMemcpy(pi0, pi0_d, sizeof(double) * V, hipMemcpyDeviceToHost);

    for (int i = 0; i < V; i++)
    {
        // printf("pi_(%d) = %lf\n", i, pi0[i]);
        pr[i] = pi0[i];
    }
}

void PersonalizedPageRank::cpu_validation(int iter)
{
    // Reset the CPU PageRank vector (uniform initialization, 1 / V for each vertex);
    std::fill(pr_golden.begin(), pr_golden.end(), 1.0 / V);

    // Do Personalized PageRank on CPU;
    auto start_tmp = clock_type::now();
    personalized_pagerank_cpu(x.data(), y.data(), val.data(), V, E, pr_golden.data(), dangling.data(), personalization_vertex, alpha, 1e-6, 100);
    auto end_tmp = clock_type::now();
    auto exec_time = chrono::duration_cast<chrono::microseconds>(end_tmp - start_tmp).count();
    std::cout << "exec time CPU=" << double(exec_time) / 1000 << " ms" << std::endl;

    // Obtain the vertices with highest PPR value;

    std::vector<std::pair<int, double>> sorted_pr_tuples = sort_pr(pr.data(), V);
    std::vector<std::pair<int, double>> sorted_pr_golden_tuples = sort_pr(pr_golden.data(), V);

    // Check how many of the correct top-20 PPR vertices are retrieved by the GPU;
    std::unordered_set<int> top_pr_indices;
    std::unordered_set<int> top_pr_golden_indices;
    int old_precision = std::cout.precision();
    std::cout.precision(4);
    int topk = std::min(V, topk_vertices);
    for (int i = 0; i < topk; i++)
    {
        int pr_id_gpu = sorted_pr_tuples[i].first;
        int pr_id_cpu = sorted_pr_golden_tuples[i].first;
        top_pr_indices.insert(pr_id_gpu);
        top_pr_golden_indices.insert(pr_id_cpu);
        if (debug)
        {
            double pr_val_gpu = sorted_pr_tuples[i].second;
            double pr_val_cpu = sorted_pr_golden_tuples[i].second;
            if (pr_id_gpu != pr_id_cpu)
            {
                std::cout << "* error in rank! (" << i << ") correct=" << pr_id_cpu << " (val=" << pr_val_cpu << "), found=" << pr_id_gpu << " (val=" << pr_val_gpu << ")" << std::endl;
            }
            else if (std::abs(sorted_pr_tuples[i].second - sorted_pr_golden_tuples[i].second) > 1e-6)
            {
                std::cout << "* error in value! (" << i << ") correct=" << pr_id_cpu << " (val=" << pr_val_cpu << "), found=" << pr_id_gpu << " (val=" << pr_val_gpu << ")" << std::endl;
            }
        }
    }
    std::cout.precision(old_precision);
    // Set intersection to find correctly retrieved vertices;
    std::vector<int> correctly_retrieved_vertices;
    set_intersection(top_pr_indices.begin(), top_pr_indices.end(), top_pr_golden_indices.begin(), top_pr_golden_indices.end(), std::back_inserter(correctly_retrieved_vertices));
    precision = double(correctly_retrieved_vertices.size()) / topk;
    if (debug)
        std::cout << "correctly retrived top-" << topk << " vertices=" << correctly_retrieved_vertices.size() << " (" << 100 * precision << "%)" << std::endl;
}

std::string PersonalizedPageRank::print_result(bool short_form)
{
    if (short_form)
    {
        return std::to_string(precision);
    }
    else
    {
        // Print the first few PageRank values (not sorted);
        std::ostringstream out;
        out.precision(3);
        out << "[";
        for (int i = 0; i < std::min(20, V); i++)
        {
            out << pr[i] << ", ";
        }
        out << "...]";
        return out.str();
    }
}

void PersonalizedPageRank::clean()
{
    // Delete any GPU data or additional CPU data;
    /*
    free(positive_residues);
    free(new_frontier);
    free(frontier);
    free(flags);
    free(pi0);
    free(residues);
    free(neighbors);
    free(neighbor_start_idx);
    free(outdegrees);
    hipFree(flags_d);
    hipFree(pi0_d);
    hipFree(residues_d);
    hipFree(neighbors_d);
    hipFree(neighbor_start_idx_d);
    */
}
