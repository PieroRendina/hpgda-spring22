#include "hip/hip_runtime.h"
// Copyright (c) 2020, 2021, NECSTLab, Politecnico di Milano. All rights reserved.

// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NECSTLab nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//  * Neither the name of Politecnico di Milano nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.

// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include <sstream>
#include "personalized_pagerank.cuh"
#include <list>
#include <vector>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
namespace chrono = std::chrono;
using clock_type = chrono::high_resolution_clock;

//////////////////////////////
//////////////////////////////
#define CHECK_KERNELCALL()                                                                \
    {                                                                                     \
        const hipError_t err = hipGetLastError();                                       \
        if (err != hipSuccess)                                                           \
        {                                                                                 \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }

#define CHECK(call)                                                                       \
    {                                                                                     \
        const hipError_t err = call;                                                     \
        if (err != hipSuccess)                                                           \
        {                                                                                 \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }

// Write GPU kernel here!
__global__ void modify_device_array_value(double *device_array, int index, double value)
{
    // residues[personalization_vertex] = 1.0;
    device_array[index] = value;
}

__global__ void update_pi0_and_r(int *frontier_d, double alpha, double *pi0_d, double *r_d, int dim_frontier)
{
    // compute the index of the vertex in the frontier
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < dim_frontier; i += blockDim.x * gridDim.x)
    {
        pi0_d[frontier_d[i]] += alpha * r_d[frontier_d[i]];
        r_d[frontier_d[i]] = 0.0;
    }
}

//  TODO change this function because it doesn't work
__global__ void compute_new_frontier(double *r_d, double rmax, bool *flags_d, int *outdegrees, double alpha, int *out_neighbors, int tot_neighbors, int * frontier, int dim_frontier, double * rsum_d, double * pi0_d)
{
    /*for(int j = blockIdx.x * blockDim.x + threadIdx.x; j < tot_neighbors ; j += blockDim.x * gridDim.x) {
        if(outdegrees[j] > 0){
            r_d[out_neighbors[j]] += (1 - alpha)*r_d[out_neighbors[j]]/outdegrees[j];
            if(r_d[out_neighbors[j]]/outdegrees[j] > rmax && flags_d[out_neighbors[j]] != true) {
                //thread syncing before updating the flags
                __syncthreads();
                flags_d[out_neighbors[j]] = true;
          }
        }else{
            r_d[out_neighbors[j]] = 0;
        }
    }*/
    for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < dim_frontier; j += blockDim.x)
    {
        int neighbor_idx = 0;
        if (j - 1 >= 0)
        {
            neighbor_idx += outdegrees[frontier[j - 1]];
        }
        int idx = neighbor_idx;
        while (idx < neighbor_idx + outdegrees[frontier[j]])
        {
            // changed out_neighbors[j] in frontier[j] 
            r_d[out_neighbors[idx]] += (1 - alpha) * r_d[frontier[j]] / outdegrees[frontier[j]];
            //printf("out neighbor considered = %d, residue = %lf, outdegree = %d\n", out_neighbors[idx], r_d[out_neighbors[idx]], outdegrees[out_neighbors[idx]]);
            
            if(outdegrees[out_neighbors[idx]] > 0) {
                if (r_d[out_neighbors[idx]]/outdegrees[out_neighbors[idx]] > rmax && flags_d[out_neighbors[idx]] != true)
                {
                    __syncthreads();
                    flags_d[out_neighbors[idx]] = true;
                }
            }
            else {
                pi0_d[out_neighbors[idx]] += alpha*r_d[out_neighbors[idx]];
                r_d[out_neighbors[idx]] = 0.0;
            }
            idx++;
        }
        // update of the rsum
        atomicAdd(&rsum_d[0], -alpha*r_d[frontier[j]]);
        //printf("value of rsum = %lf\n", rsum_d[0]);
    }
}


__global__ void random_walks(double random_walks_factor, double w, double * pi0_d, double * residues_d, int tot_nodes, int * starting_nodes, int * outdegrees, int * neighbor_start_idx_d, double alpha)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState state;
    hiprand_init(0, i, 0, &state);
    // iterate through all the starting nodes 
    for ( ; i < tot_nodes; i += blockDim.x * gridDim.x) 
    {
        // compute the number of walks to do
        //double wi = ceil(residues_d[starting_nodes[i]] * w/rsum_d[0]);
        double wi = ceil(outdegrees[starting_nodes[i]]*random_walks_factor);
        printf("\nnumber of walks to do = %lf at iteration %d\n", wi, i);
        printf("starting node: %d, outdegree: %d\n", starting_nodes[i], outdegrees[starting_nodes[i]]);
        // double ai = (residues_d[i] * w) / (rsum * wi);
        // compute monte carlo
        // pi0_d[i] += ai*rsum/w
        for (int walks_done = 0; walks_done < wi; walks_done++)
        {
            float flip = (float)hiprand_uniform(&state);
            int current_node = starting_nodes[i];
            while(outdegrees[current_node] > 0 && flip < (1-alpha)) {
                int decision_interval = neighbor_start_idx_d[current_node + 1] - neighbor_start_idx_d[current_node];
                int neighbor_chosen = ceil(decision_interval*hiprand_uniform(&state) + neighbor_start_idx_d[current_node]);
                current_node = neighbor_chosen;
                flip = hiprand_uniform(&state);
            }
            atomicAdd(&pi0_d[current_node], residues_d[starting_nodes[i]]/wi);
        }
    }
}

__global__ void perform_random_walks(int n_walks, int starting_node, int * outdegrees, int * neighbor_start_idx_d, double alpha, double * pi0_d, double * residues_d) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState state;
    hiprand_init(0, i, 0, &state);
    for (; i < n_walks; i += blockDim.x * gridDim.x) {
            float flip = (float)hiprand_uniform(&state);
            int current_node = starting_node;
            while(outdegrees[current_node] > 0 && flip < (1-alpha)) {
                int decision_interval = neighbor_start_idx_d[current_node + 1] - neighbor_start_idx_d[current_node];
                int neighbor_chosen = ceil(decision_interval*hiprand_uniform(&state) + neighbor_start_idx_d[current_node]);
                current_node = neighbor_chosen;
                flip = hiprand_uniform(&state);
            }
            atomicAdd(&pi0_d[current_node], residues_d[starting_node]/n_walks);
    }
}


//////////////////////////////
/* CPU function */
//////////////////////////////

void PersonalizedPageRank::initialize_outdegrees() {
    for (int i = 0; i < V; i++)
    {   
        // printf("node in the frontier = %d\n", frontier[i]);
        int start_idx = neighbor_start_idx[i];
        int end_idx = neighbor_start_idx[i + 1];
        // outdegree computation
        outdegrees[i] = end_idx - start_idx;
        // the node is dropped from the frontier
        flags[frontier[i]] = false;
    }

    for(int i = 0; i < V; i++) {
        printf("Outdegree of node %d = %d\n", i, outdegrees[i]);
    }
}


void PersonalizedPageRank::generate_random_walks(int count_positive_residues) {
    for(int i = 0; i < count_positive_residues; i++) {
        int walks_per_node = ceil(outdegrees[positive_residues[i]]*random_walks_factor);
        int n_blocks = 1;
        int n_threads = 32;
        printf("Starting node = %d, number of random_walks = %d\n", positive_residues[i], walks_per_node);
        perform_random_walks<<<n_blocks, n_threads>>>(walks_per_node, positive_residues_d[i], outdegrees, neighbor_start_idx_d, alpha, pi0_d, residues_d);
        CHECK(hipDeviceSynchronize());
    }
}



void PersonalizedPageRank::update_frontiers()
{

    /* --- Compute the number of neighbors and drop the nodes from the frontier --- */
    // allocate the vector to store the degree of each node in the frontier
    int tot_neighbors = 0;
    //int *outdegrees;
    /* 
    one outdegree for each member of the frontier 
    err = hipMallocManaged(&outdegrees, sizeof(int) * dim_frontier);
    printf("\nDim frontier = %d\n", dim_frontier);
    */
    for (int i = 0; i < dim_frontier; i++)
    {
        
        tot_neighbors += outdegrees[frontier[i]];
        // the node is dropped from the frontier
        flags[frontier[i]] = false;
    }
    printf("Number of outneighbors = %d\n", tot_neighbors);
    /* --- Add the neighbours to be considered in the vector out_neighbors --- */
    int *out_neighbors;
    /* all the neighbors to be considered */
    err = hipMallocManaged(&out_neighbors, sizeof(int) * tot_neighbors);
    int counter = 0;
    for (int i = 0; i < dim_frontier; i++)
    {
        int start_idx = neighbor_start_idx[frontier[i]];
        int end_idx = neighbor_start_idx[frontier[i] + 1];
        for (int j = start_idx; j < end_idx; j++)
        {
            out_neighbors[counter] = neighbors[j];
            counter += 1;
        }
    }
    /* --- Update of the frontier --- */
    // int new_frontier_dim = 3*dim_frontier;
    // int new_frontier_dim = 30 * dim_frontier;
    // int *new_frontier;
    /* --- For simplicity I allocate a frontier that is long the number of vertex --- */
    err = hipMallocManaged(&new_frontier, sizeof(int) * V);
    // int *new_frontier = (int *)malloc(new_frontier_dim * sizeof(int));

    /* For each out_neighbor updates the residue and checks if it has to be added to the frontier */
    // compute_new_frontier<<<ceil(tot_neighbors/1024)+1, ceil(tot_neighbors/ceil(tot_neighbors/1024))+1>>>(residues_d, rmax, flags_d, outdegrees, alpha, out_neighbors, tot_neighbors);

    int n_blocks = ceil(tot_neighbors / 1024) + 1;
    int n_threads = ceil(tot_neighbors / n_blocks) + 1;
    hipMemcpy(flags_d, flags, sizeof(bool) * V, hipMemcpyHostToDevice);
    compute_new_frontier<<<n_blocks,n_threads>>>(residues_d, rmax, flags_d, outdegrees, alpha, out_neighbors, tot_neighbors, frontier, dim_frontier, rsum_d, pi0_d);
    CHECK(hipDeviceSynchronize());
    hipMemcpy(flags, flags_d, sizeof(bool) * V, hipMemcpyDeviceToHost);

    int idx_frontier = 0;
    for (int i = 0; i < V; i++)
    {
        if (flags[i] == true)
        {
            /*
            if(idx_frontier >= new_frontier_dim){
              new_frontier_dim = 3 * new_frontier_dim;
              new_frontier = (int *)realloc(new_frontier, new_frontier_dim * sizeof(int));
              printf("--- realloc done --- \n");
            }*/
            new_frontier[idx_frontier] = i;
            idx_frontier++;
        }
    }

    /* -- Check the flag -- */

    // hipFree(frontier);
    // frontier = new_frontier;
    new_dim_frontier = idx_frontier;
    /*std::cout << "----- Updated frontier -----\n";
    for (int i = 0; i < new_dim_frontier; i++)
    {
        std::cout << new_frontier[i] << " ";
    }*/
}

__global__ void initialize_csr_parallel(int * x, int * y, int V, int E, int ** out_neighbors, int * outdegrees) {
    for (int index = threadIdx.x + blockDim.x * blockIdx.x; index < V; index += blockDim.x) {
        int n_neighbors = 0;
        for(int j = 0; j < E; j++) {
            if(y[j] == index) {
                n_neighbors++;
            }
        }
        out_neighbors[index] = (int*)malloc(sizeof(int)*n_neighbors);
        outdegrees[index] = n_neighbors;
        int curr_idx = 0;
        for(int j = 0; j < E; j++) {
            if(y[j] == index) {
                out_neighbors[index][curr_idx] = x[j];
                curr_idx++;
            }
        }
    }
}

__global__ void print_neighbors(int ** out_neighbors, int * outdegrees, int V) {
    for(int i = 0; i < V; i++) {
        for(int j = 0; j < outdegrees[i]; j++) {
            printf("%d ", out_neighbors[i][j]);
        }
    }
}


void PersonalizedPageRank::initialize_csr()
{
    // allocate a vector containing the index of the starting neighbor
    neighbor_start_idx = (int *)malloc((V + 1) * sizeof(int));
    neighbors = (int *)malloc(E * sizeof(int));

    int curr_neighbor = 0;
    int curr_neighbor_start_idx = 1;
    neighbor_start_idx[0] = 0;

    for (int i = 0; i < V; i++)
    {
        //auto start = std::chrono::system_clock::now();
        for (int j = 0; j < E; j++)
        {
            if (y[j] == i)
            {
                neighbors[curr_neighbor] = x[j];
                curr_neighbor++;
            }
        }
        neighbor_start_idx[curr_neighbor_start_idx] = curr_neighbor;
        outdegrees[i] = neighbor_start_idx[curr_neighbor_start_idx] - neighbor_start_idx[curr_neighbor_start_idx - 1];
        curr_neighbor_start_idx++;
        /*auto end = std::chrono::system_clock::now();
        std::chrono::duration<double> elapsed_seconds = end-start;
        printf("Iteration done in %f\n", elapsed_seconds.count());*/
    }

    printf("\n --- Finished CSR initialization --- \n");

    
    /*std::cout << "----- Outdegrees -----\n";
        for (int i = 0; i < V ; i++)
        {
            std::cout << outdegrees[i] << " ";
        }
        std::cout << "\n----- Neighbours -----\n";
        for (int i = 0; i < V + 1; i++)
        {
            std::cout << neighbor_start_idx[i] << " ";
        }
    */
    /*    std::cout << "\n----- Neighbours -----\n";
        for (int i = 0; i < E; i++)
        {
           std::cout << neighbors[i] << " ";
        }
    */
}

// CPU Utility functions;

// Read the input graph and initialize it;
void PersonalizedPageRank::initialize_graph()
{
    // Read the graph from an MTX file;
    int num_rows = 0;
    int num_columns = 0;
    read_mtx(graph_file_path.c_str(), &x, &y, &val,
             &num_rows, &num_columns, &E, // Store the number of vertices (row and columns must be the same value), and edges;
             true,                        // If true, read edges TRANSPOSED, i.e. edge (2, 3) is loaded as (3, 2). We set this true as it simplifies the PPR computation;
             false,                       // If true, read the third column of the matrix file. If false, set all values to 1 (this is what you want when reading a graph topology);
             debug,
             false, // MTX files use indices starting from 1. If for whatever reason your MTX files uses indices that start from 0, set zero_indexed_file=true;
             true   // If true, sort the edges in (x, y) order. If you have a sorted MTX file, turn this to false to make loading faster;
    );
    if (num_rows != num_columns)
    {
        if (debug)
            std::cout << "error, the matrix is not squared, rows=" << num_rows << ", columns=" << num_columns << std::endl;
        exit(-1);
    }
    else
    {
        V = num_rows;
    }
    if (debug)
        std::cout << "loaded graph, |V|=" << V << ", |E|=" << E << std::endl;

    // Compute the dangling vector. A vertex is not dangling if it has at least 1 outgoing edge;
    dangling.resize(V);
    std::fill(dangling.begin(), dangling.end(), 1); // Initially assume all vertices to be dangling;
    for (int i = 0; i < E; i++)
    {
        // Ignore self-loops, a vertex is still dangling if it has only self-loops;
        if (x[i] != y[i])
            dangling[y[i]] = 0;
    }
    // Initialize the CPU PageRank vector;
    pr.resize(V);
    pr_golden.resize(V);
    // Initialize the value vector of the graph (1 / outdegree of each vertex).
    // Count how many edges start in each vertex (here, the source vertex is y as the matrix is transposed);
    int *outdegree = (int *)calloc(V, sizeof(int));
    for (int i = 0; i < E; i++)
    {
        outdegree[y[i]]++;
    }
    // Divide each edge value by the outdegree of the source vertex;
    for (int i = 0; i < E; i++)
    {
        val[i] = 1.0 / outdegree[y[i]];
    }
    free(outdegree);
}

//////////////////////////////
//////////////////////////////

// Allocate data on the CPU and GPU;

void PersonalizedPageRank::alloc()
{
    // Load the input graph and preprocess it;
    //read_from_file();
    initialize_graph();
    // allocate the mask to store the status of the nodes in the frontier (all false by default)
    flags = (bool *)calloc(V, sizeof(bool));
    // at the beginning the frontier contains just the personalization vertex
    // frontier = (int *)malloc(sizeof(int));
    err = hipMallocManaged(&frontier, sizeof(int));
    err = hipMallocManaged(&outdegrees, sizeof(int)*V);

    // Attempt of parallelizing the CSR construction 
    err = hipMallocManaged(&out_neighbors, sizeof(int*)*V);
    err = hipMallocManaged(&x_d, sizeof(int)*E);
    err = hipMallocManaged(&y_d, sizeof(int)*E);


    for(int i = 0; i < E; i++) {
        x_d[i] = x[i];
        y_d[i] = y[i];
    }


    initialize_csr();

    // Allocate any GPU data here;
    // TODO!
    err = hipMalloc(&neighbor_start_idx_d, sizeof(int) * (V + 1));
    err = hipMalloc(&neighbors_d, sizeof(int) * E);
    err = hipMalloc(&pi0_d, sizeof(double) * V);
    err = hipMalloc(&residues_d, sizeof(double) * V);
    err = hipMalloc(&flags_d, sizeof(bool) * V);
}



// Initialize data;

void PersonalizedPageRank::init()
{
    // Do any additional CPU or GPU setup here;
    // TODO!

    // Compute Rmax
    threshold = 1.0 / V; // should be O(1/n) but i don't know yet which is the best value
    rmax = (convergence_threshold / sqrt(E)) * sqrt(threshold / (((2.0 * convergence_threshold / 3.0) + 2.0) * (log(2.0 / failure_probability))));
    std::cout << "rmax = " << rmax << '\n'; // It seems really small
    random_walks_factor = rmax*((2*convergence_threshold/3+2)*log(2*V*log(V)/failure_probability))/(threshold*convergence_threshold*convergence_threshold);

    hipMemset(residues_d, 0.0, sizeof(double) * V);
    hipMemset(pi0_d, 0.0, sizeof(double) * V);
    hipMemcpy(neighbor_start_idx_d, neighbor_start_idx, sizeof(int) * (V + 1), hipMemcpyHostToDevice);
    hipMemcpy(neighbors_d, neighbors, sizeof(int) * E, hipMemcpyHostToDevice);
    hipMemcpy(flags_d, flags, sizeof(bool) * V, hipMemcpyHostToDevice);
}

/*void PersonalizedPageRank::alloc() {
  for(int i = 0; i < 10; i++) {
    printf("%d) neighbor start index = %d, outdegree = %d\n", i, neighbor_start_idx[i], outdegrees[i]);
  }
}*/

// Reset the state of the computation after every iteration.
// Reset the result, and transfer data to the GPU if necessary;
void PersonalizedPageRank::reset()
{
    // Reset the PageRank vector (uniform initialization, 1 / V for each vertex);
    std::fill(pr.begin(), pr.end(), 1.0 / V);
    // Generate a new personalization vertex for this iteration;
    personalization_vertex = rand() % V;
    //personalization_vertex = 6016;
    if (debug)
        std::cout << "personalization vertex=" << personalization_vertex << std::endl;

    // Do any GPU reset here, and also transfer data to the GPU;
    // TODO!

    hipMemset(residues_d, 0.0, sizeof(double) * V);
    hipMemset(pi0_d, 0.0, sizeof(double) * V);
    hipMemcpy(flags_d, flags, sizeof(bool) * V, hipMemcpyHostToDevice);

    /*
    // Free old variables
    free(positive_residues);
    free(new_frontier);
    free(frontier);
    free(residues);
    */
}

void PersonalizedPageRank::execute(int iter)
{
    // Do the GPU computation here, and also transfer results to the CPU;
    // TODO! (and save the GPU PPR values into the "pr" array)
    /*while(frontier.size() > 0) {


    }*/
    // This should be moved to the execute stage.
    err = hipMallocManaged(&rsum_d, sizeof(double));
    rsum_d[0] = 1.0;
    dim_frontier = 1;
    frontier[0] = personalization_vertex;
    flags[personalization_vertex] = true;
    modify_device_array_value<<<1, 1>>>(residues_d, personalization_vertex, 1.0);
    CHECK(hipDeviceSynchronize());
    // int dim_frontier_old; // used to update pi0 and r
    // int ma = 0;
    while (dim_frontier > 0)
    {
        // printf("\n----> iteration %d\n", ma+1);
        // ma++;
        int n_blocks = ceil(dim_frontier / 1024) + 1;
        int n_threads = ceil(dim_frontier / n_blocks) + 1;
        // CHECK(hipDeviceSynchronize());
        // dim_frontier_old = dim_frontier;
        update_frontiers();
        CHECK(hipDeviceSynchronize());
        update_pi0_and_r<<<n_blocks, n_threads>>>(frontier, alpha, pi0_d, residues_d, dim_frontier);
        CHECK(hipDeviceSynchronize());
        frontier = new_frontier;
        dim_frontier = new_dim_frontier;
    }
    pi0 = (double *)malloc(sizeof(double) * V);
    hipMemcpy(pi0, pi0_d, sizeof(double) * V, hipMemcpyDeviceToHost);

    // New part --->
    residues = (double *)malloc(sizeof(double) * V);
    hipMemcpy(residues, residues_d, sizeof(double) * V, hipMemcpyDeviceToHost);

    int count_positive_residues = 0;
    positive_residues = (int *)malloc(sizeof(int) * V);

    // compute rsum and save in an array all the nodes with positive residues
    for (int i = 0; i < V; i++)
    {
        // stores just the nodes whose outdegree is positive
        if (residues[i] > 0 && outdegrees[i] > 0)
        {
            //rsum += residues[i];
            positive_residues[count_positive_residues] = i;
            count_positive_residues++;
        }
    }

    /* adjust the size
    positive_residues = realloc(positive_residues, count_positive_residues);
    printf("--- Positive residues after realloc ---\n");
    for(int i = 0; i < count_positive_residues; i++) {
        printf("node with positive residue: %d\n", positive_residues[i]);
    }*/
    err = hipMallocManaged(&positive_residues_d, sizeof(int)*count_positive_residues);
    for(int i = 0; i < count_positive_residues; i++) {
        positive_residues_d[i] = positive_residues[i];
    }

    //free(positive_residues);

    //hipMemcpy(positive_residues_d, positive_residues, sizeof(int)*count_positive_residues, hipMemcpyHostToDevice);

    //w = rsum_d[0] * ((2 * threshold / 3 + 2) * log(2.0 / failure_probability) / (convergence_threshold * threshold * threshold));
    //printf("Number of random walks to do = %lf\n", w);
    // for every node in positive_residue instantiate a thread on the GPU and compute the random walks
    printf("Number of positive residues = %d\n", count_positive_residues);
    //hipMemcpy(pi0_d, pi0, sizeof(double) * V, hipMemcpyHostToDevice); // i'm not sure that this step is necessary
    //hipMemcpy(residues_d, residues, sizeof(double) * V, hipMemcpyHostToDevice);

    /* --- Parallelizing for each node, random walks done sequentially --- 
    int n_blocks = ceil(count_positive_residues / 1024) + 1;
    int n_threads = ceil(count_positive_residues / n_blocks) + 1;
    random_walks<<<n_blocks, n_threads>>>(random_walks_factor, w, pi0_d, residues_d, count_positive_residues, positive_residues_d, outdegrees, neighbor_start_idx_d, alpha);
    CHECK(hipDeviceSynchronize()); */

    /* --- Parallelizing random walks per node --- */
    generate_random_walks(count_positive_residues);


    // End new part <----

    // printf("\n--- Estimated pi ---\n");
    for (int i = 0; i < V; i++)
    {
        // printf("pi_(%d) = %lf\n", i, pi0[i]);
        pr[i] = pi0[i];
    }

}

void PersonalizedPageRank::cpu_validation(int iter)
{
    // Reset the CPU PageRank vector (uniform initialization, 1 / V for each vertex);
    std::fill(pr_golden.begin(), pr_golden.end(), 1.0 / V);

    // Do Personalized PageRank on CPU;
    auto start_tmp = clock_type::now();
    personalized_pagerank_cpu(x.data(), y.data(), val.data(), V, E, pr_golden.data(), dangling.data(), personalization_vertex, alpha, 1e-6, 100);
    auto end_tmp = clock_type::now();
    auto exec_time = chrono::duration_cast<chrono::microseconds>(end_tmp - start_tmp).count();
    std::cout << "exec time CPU=" << double(exec_time) / 1000 << " ms" << std::endl;

    // Obtain the vertices with highest PPR value;
    std::vector<std::pair<int, double>> sorted_pr_tuples = sort_pr(pr.data(), V);
    std::vector<std::pair<int, double>> sorted_pr_golden_tuples = sort_pr(pr_golden.data(), V);

    // Check how many of the correct top-20 PPR vertices are retrieved by the GPU;
    std::unordered_set<int> top_pr_indices;
    std::unordered_set<int> top_pr_golden_indices;
    int old_precision = std::cout.precision();
    std::cout.precision(4);
    int topk = std::min(V, topk_vertices);
    for (int i = 0; i < topk; i++)
    {
        int pr_id_gpu = sorted_pr_tuples[i].first;
        int pr_id_cpu = sorted_pr_golden_tuples[i].first;
        top_pr_indices.insert(pr_id_gpu);
        top_pr_golden_indices.insert(pr_id_cpu);
        if (debug)
        {
            double pr_val_gpu = sorted_pr_tuples[i].second;
            double pr_val_cpu = sorted_pr_golden_tuples[i].second;
            if (pr_id_gpu != pr_id_cpu)
            {
                std::cout << "* error in rank! (" << i << ") correct=" << pr_id_cpu << " (val=" << pr_val_cpu << "), found=" << pr_id_gpu << " (val=" << pr_val_gpu << ")" << std::endl;
            }
            else if (std::abs(sorted_pr_tuples[i].second - sorted_pr_golden_tuples[i].second) > 1e-6)
            {
                std::cout << "* error in value! (" << i << ") correct=" << pr_id_cpu << " (val=" << pr_val_cpu << "), found=" << pr_id_gpu << " (val=" << pr_val_gpu << ")" << std::endl;
            }
        }
    }
    std::cout.precision(old_precision);
    // Set intersection to find correctly retrieved vertices;
    std::vector<int> correctly_retrieved_vertices;
    set_intersection(top_pr_indices.begin(), top_pr_indices.end(), top_pr_golden_indices.begin(), top_pr_golden_indices.end(), std::back_inserter(correctly_retrieved_vertices));
    precision = double(correctly_retrieved_vertices.size()) / topk;
    if (debug)
        std::cout << "correctly retrived top-" << topk << " vertices=" << correctly_retrieved_vertices.size() << " (" << 100 * precision << "%)" << std::endl;
}

std::string PersonalizedPageRank::print_result(bool short_form)
{
    if (short_form)
    {
        return std::to_string(precision);
    }
    else
    {
        // Print the first few PageRank values (not sorted);
        std::ostringstream out;
        out.precision(3);
        out << "[";
        for (int i = 0; i < std::min(20, V); i++)
        {
            out << pr[i] << ", ";
        }
        out << "...]";
        return out.str();
    }
}

void PersonalizedPageRank::clean()
{
    // Delete any GPU data or additional CPU data;
    // TODO!
}




// Reader functions
void PersonalizedPageRank::read_from_file()
{
    int read_idx = 0;
    int read_out = 0;
    int count = 0;

    FILE *f;
    char line[256];
    f = fopen("/content/drive/MyDrive/hpgda-spring22-main/cuda_CONTEST/data/demo.txt", "r");

    fgets(line, 256, f);
    fgets(line, 256, f);

    neighbors = (int *)malloc(atoi(line) * sizeof(int));
    printf("Neighbours number: %d\n", atoi(line));
    while (fgets(line, 256, f) != NULL)
    {
        if (line[0] != '#' && read_idx == 0 && read_out == 0)
        { // read neighbords
            neighbors[count] = atoi(line);
            count++;
        }
        else if (line[0] == '#' && read_idx == 0 && read_out == 0)
        { // end the reading of the neighbors and I will start with the idx
            fgets(line, 256, f);
            neighbor_start_idx = (int *)malloc(atoi(line) * sizeof(int));
            printf("Index number: %d\n", atoi(line));
            read_idx = 1;
            count = 0;
        }
        else if (line[0] != '#' && read_idx == 1 && read_out == 0)
        { // read index
            neighbor_start_idx[count] = atoi(line);
            count++;
        }
        else if (line[0] == '#' && read_idx == 1 && read_out == 0)
        { // end the reading of the indexes and I will start with the out
            fgets(line, 256, f);
            outdegrees = (int *)malloc(atoi(line) * sizeof(int));
            printf("Outdegree number: %d\n", atoi(line));
            read_out = 1;
            count = 0;
        }
        else
        { // read out
            outdegrees[count] = atoi(line);
            count++;
        }
    }
}
